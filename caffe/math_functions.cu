// Copyright 2013 Yangqing Jia

#include <cmath>
#include <cstdlib>
#include <cstring>

#include "common.cuh"
#include "math_functions.cuh"

//template <>
//void caffe_gpu_gemm<float>(const CBLAS_TRANSPOSE TransA,
//    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
//    const float alpha, const float* A, const float* B, const float beta,
//    float* C) {
//  // Note that cublas follows fortran order.
//  int lda = (TransA == CblasNoTrans) ? K : M;
//  int ldb = (TransB == CblasNoTrans) ? N : K;
//  hipblasOperation_t cuTransA =
//      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
//  hipblasOperation_t cuTransB =
//      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
//  CUBLAS_CHECK(hipblasSgemm(Caffe::cublas_handle(), cuTransB, cuTransA,
//      N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
//}
//
//template <>
//void caffe_gpu_gemm<double>(const CBLAS_TRANSPOSE TransA,
//    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
//    const double alpha, const double* A, const double* B, const double beta,
//    double* C) {
//  // Note that cublas follows fortran order.
//  int lda = (TransA == CblasNoTrans) ? K : M;
//  int ldb = (TransB == CblasNoTrans) ? N : K;
//  hipblasOperation_t cuTransA =
//      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
//  hipblasOperation_t cuTransB =
//      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
//  CUBLAS_CHECK(hipblasDgemm(Caffe::cublas_handle(), cuTransB, cuTransA,
//      N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
//}
