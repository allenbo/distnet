#include "hip/hip_runtime.h"
// Copyright 2013 Yangqing Jia

#include <cstdio>
#include <ctime>
#include <sys/types.h>
#include <unistd.h>

#include "common.hpp"

namespace caffe {

long cluster_seedgen(void) {
  long s, seed, pid;
  pid = getpid();
  s = time(NULL);
  seed = abs(((s * 181) * ((pid - 83) * 359)) % 104729);
  return seed;
}

}  // namespace caffe
